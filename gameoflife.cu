#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCK 16
#define ITERATIONS 10


__device__
void getCellStatusPar(int* val, int matrix[BLOCK][BLOCK], int row, int column) {
        *val = row >= 0 && row < BLOCK && column >= 0 && column < BLOCK ?
                        matrix[row][column] : 0;
}

__global__
void parallelGOL(int* matrix, int m, int n) {
        __shared__ int block[BLOCK][BLOCK];
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        int result, it, pos = i * n + j, val = 0;
        int coordY = threadIdx.y, coordX = threadIdx.x;

        if (pos < m * n) {
                for(it = 0; it < ITERATIONS; ++it){
                        block[coordY][coordX] = matrix[pos];//copying to shared memory
                        __syncthreads();

                        getCellStatusPar(&val, block, coordY - 1, coordX - 1); //UP LEFT
                        result = val;

                        getCellStatusPar(&val, block, coordY, coordX - 1); //LEFT
                        result += val;
                        getCellStatusPar(&val, block, coordY + 1, coordX - 1); //BOTTOM LEFT
                        result += val;

                        getCellStatusPar(&val, block, coordY - 1, coordX); //UP
                        result += val;
                        getCellStatusPar(&val, block, coordY + 1, coordX); //DOWN
                        result += val;

                        getCellStatusPar(&val, block, coordY - 1, coordX + 1); //UP RIGHT
                        result += val;
                        getCellStatusPar(&val, block, coordY, coordX + 1); //RIGHT
                        result += val;
                        getCellStatusPar(&val, block, coordY + 1, coordX + 1); //BOTTOM RIGHT
                        result += val;


                        if (result == 2 && matrix[pos] == 1)
                                matrix[pos] = 1;
                        else if (result == 3)
                                matrix[pos] = 1;
                        else
                                matrix[pos] = 0;
                        __syncthreads(); //synchronization step after writing and before updating "block"
                }
        }
}

int main(int argc, char *argv[]) {
        if (argc < 3) {
                printf("No enough arguments.");
                return -1;
        }

        srand (time(NULL));

        int m = atoi(argv[1]);//number of blocks in Y axis!!!!
        int n = atoi(argv[2]);//number of blocks in YXaxis!!!!

        int* matrix = (int*) malloc(BLOCK * BLOCK * m * n * sizeof(int));

        int realM = m * BLOCK;
        int realN = n * BLOCK;

        int i;
        for (i = 0; i < realM * realN; ++i)
                matrix[i] = rand() % 2; //0 = dead, 1 = alive

        //starting parallel execution
        clock_t t = clock();
        int *d_matrix;


        //allocating memory
        hipMalloc(&d_matrix, realN * realM * sizeof(int));

        //copying memory
        hipMemcpy(d_matrix, matrix, realN * realM * sizeof(int), hipMemcpyHostToDevice);

        //dimensions
        dim3 threadblock(BLOCK, BLOCK); //16 * 16 = 256 in total
        dim3 grid(1 + realN / threadblock.x, 1 + realM / threadblock.y);

        //executing the function
        parallelGOL<<<grid, threadblock>>>(d_matrix, realM, realN);
        hipDeviceSynchronize();


        //once the function has been called I copy the result in matrix
        hipMemcpy(matrix, d_matrix, realN * realM * sizeof(int), hipMemcpyDeviceToHost);

        double parallelExecutionTime = ((double) (clock() - t))
                        / ((double) (CLOCKS_PER_SEC));

        printf("%d;%f\n", realM, parallelExecutionTime);

        //free resources
        hipFree(d_matrix);
        free(matrix);

        return 0;
}
